
#include <hip/hip_runtime.h>
const char program_source[] = R"%%%(

#include "cuda_helper.h"
#include "ethash_cuda_miner_kernel.h"
#include "ethash_cuda_miner_kernel_globals.h"
#include "fnv.cuh"

#define copy(dst, src, count)        \
    for (int i = 0; i != count; ++i) \
    {                                \
        (dst)[i] = (src)[i];         \
    }

#include "dagger_shuffled.cuh"
#include "keccak.cuh"

#define __ASSEMBLER__
#define __extension__
#include <stdint.h>

__global__ void ethash_search(volatile Search_results* g_output, uint64_t start_nonce)
{
    uint32_t const gid = blockIdx.x * blockDim.x + threadIdx.x;
    uint2 mix[4];
    if (compute_hash(start_nonce + gid, mix))
        return;
    uint32_t index = atomicInc((uint32_t*)&g_output->count, 0xffffffff);
    if (index >= MAX_SEARCH_RESULTS)
        return;
    g_output->result[index].gid = gid;
    g_output->result[index].mix[0] = mix[0].x;
    g_output->result[index].mix[1] = mix[0].y;
    g_output->result[index].mix[2] = mix[1].x;
    g_output->result[index].mix[3] = mix[1].y;
    g_output->result[index].mix[4] = mix[2].x;
    g_output->result[index].mix[5] = mix[2].y;
    g_output->result[index].mix[6] = mix[3].x;
    g_output->result[index].mix[7] = mix[3].y;
}

void run_ethash_search(uint32_t gridSize, uint32_t blockSize, chipStream_tstream,
    volatile Search_results* g_output, uint64_t start_nonce)
{
    ethash_search<<<gridSize, blockSize, 0, stream>>>(g_output, start_nonce);
    CUDA_SAFE_CALL(cudaGetLastError());
}

#define ETHASH_DATASET_PARENTS 256
#define NODE_WORDS (64 / 4)


__global__ void ethash_calculate_dag_item(uint32_t start)
{
    uint32_t const node_index = start + blockIdx.x * blockDim.x + threadIdx.x;
    if (((node_index >> 1) & (~1)) >= d_dag_size)
        return;
    union {
       hash128_t dag_node;
       uint2 dag_node_mem[25];
    };
    copy(dag_node.uint4s, d_light[node_index % d_light_size].uint4s, 4);
    dag_node.words[0] ^= node_index;
    SHA3_512(dag_node_mem);

    const int thread_id = threadIdx.x & 3;

    for (uint32_t i = 0; i != ETHASH_DATASET_PARENTS; ++i)
    {
        uint32_t parent_index = fnv(node_index ^ i, dag_node.words[i % NODE_WORDS]) % d_light_size;
        for (uint32_t t = 0; t < 4; t++)
        {
            uint32_t shuffle_index = SHFL(parent_index, t, 4);

            uint4 p4 = d_light[shuffle_index].uint4s[thread_id];
            for (int w = 0; w < 4; w++)
            {
                uint4 s4 = make_uint4(SHFL(p4.x, w, 4), SHFL(p4.y, w, 4), SHFL(p4.z, w, 4), SHFL(p4.w, w, 4));
                if (t == thread_id)
                {
                    dag_node.uint4s[w] = fnv4(dag_node.uint4s[w], s4);
                }
            }
        }
    }
    SHA3_512(dag_node_mem);
    hash64_t* dag_nodes = (hash64_t*)d_dag;
    copy(dag_nodes[node_index].uint4s, dag_node.uint4s, 4);
}

void ethash_generate_dag(
    uint64_t dag_size, uint32_t gridSize, uint32_t blockSize, chipStream_tstream)
{
    const uint32_t work = (uint32_t)(dag_size / sizeof(hash64_t));
    const uint32_t run = gridSize * blockSize;

    uint32_t base;
    for (base = 0; base <= work - run; base += run)
    {
        ethash_calculate_dag_item<<<gridSize, blockSize, 0, stream>>>(base);
        CUDA_SAFE_CALL(cudaDeviceSynchronize());
    }
    if (base < work)
    {
        uint32_t lastGrid = work - base;
        lastGrid = (lastGrid + blockSize - 1) / blockSize;
        ethash_calculate_dag_item<<<lastGrid, blockSize, 0, stream>>>(base);
        CUDA_SAFE_CALL(cudaDeviceSynchronize());
    }
    CUDA_SAFE_CALL(cudaGetLastError());
}

void set_constants(hash128_t* _dag, uint32_t _dag_size, hash64_t* _light, uint32_t _light_size)
{
    CUDA_SAFE_CALL(cudaMemcpyToSymbol(d_dag, &_dag, sizeof(hash128_t*)));
    CUDA_SAFE_CALL(cudaMemcpyToSymbol(d_dag_size, &_dag_size, sizeof(uint32_t)));
    CUDA_SAFE_CALL(cudaMemcpyToSymbol(d_light, &_light, sizeof(hash64_t*)));
    CUDA_SAFE_CALL(cudaMemcpyToSymbol(d_light_size, &_light_size, sizeof(uint32_t)));
}

void get_constants(hash128_t** _dag, uint32_t* _dag_size, hash64_t** _light, uint32_t* _light_size)
{
    /*
       Using the direct address of the targets did not work.
       So I've to read first into local variables when using cudaMemcpyFromSymbol()
    */
    if (_dag)
    {
        hash128_t* _d;
        CUDA_SAFE_CALL(cudaMemcpyFromSymbol(&_d, d_dag, sizeof(hash128_t*)));
        *_dag = _d;
    }
    if (_dag_size)
    {
        uint32_t _ds;
        CUDA_SAFE_CALL(cudaMemcpyFromSymbol(&_ds, d_dag_size, sizeof(uint32_t)));
        *_dag_size = _ds;
    }
    if (_light)
    {
        hash64_t* _l;
        CUDA_SAFE_CALL(cudaMemcpyFromSymbol(&_l, d_light, sizeof(hash64_t*)));
        *_light = _l;
    }
    if (_light_size)
    {
        uint32_t _ls;
        CUDA_SAFE_CALL(cudaMemcpyFromSymbol(&_ls, d_light_size, sizeof(uint32_t)));
        *_light_size = _ls;
    }
}

void set_header(hash32_t _header)
{
    CUDA_SAFE_CALL(cudaMemcpyToSymbol(d_header, &_header, sizeof(hash32_t)));
}

void set_target(uint64_t _target)
{
    CUDA_SAFE_CALL(cudaMemcpyToSymbol(d_target, &_target, sizeof(uint64_t)));
}
)%%%";
